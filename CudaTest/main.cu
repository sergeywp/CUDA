#include "hip/hip_runtime.h"
#include ""
#include <iostream>

using namespace std;

int main(){
	hipDeviceProp_t prop; 

	int counted;

	hipGetDeviceCount(&counted);

	for(int i=0; i<counted; i++){ 
		hipGetDeviceProperties(&prop,i);
		cout<<"---Some Information for the Device---"<<endl;
		cout<<"Name : " << prop.name << endl;
		cout<<"Compute capability : " << prop.major << "."<< prop.minor << endl;
		cout<<"Clock Rate : " << prop.clockRate << endl;
	}
}